#include "hip/hip_runtime.h"
/*
 * k-Means clustering algorithm
 *
 * CUDA version
 *
 * Parallel computing (Degree in Computer Engineering)
 * 2022/2023
 *
 * Version: 1.0
 *
 * (c) 2022 Diego García-Álvarez, Arturo Gonzalez-Escribano
 * Grupo Trasgo, Universidad de Valladolid (Spain)
 *
 * This work is licensed under a Creative Commons Attribution-ShareAlike 4.0 International License.
 * https://creativecommons.org/licenses/by-sa/4.0/
 */

 #include <stdio.h>
 #include <stdlib.h>
 #include <ctype.h>
 #include <math.h>
 #include <time.h>
 #include <string.h>
 #include <float.h>
 #include <kmeans.h>
 //#include <hip/hip_runtime.h>
 
 #define MAXCAD 200
 
 //Macros
 #define MIN(a,b) ((a) < (b) ? (a) : (b))
 #define MAX(a,b) ((a) > (b) ? (a) : (b))
 
 /*
  * Macros to show errors when calling a CUDA library function,
  * or after launching a kernel
  */
 #define CHECK_CUDA_CALL( a )	{ \
     hipError_t ok = a; \
     if ( ok != hipSuccess ) \
         fprintf(stderr, "-- Error CUDA call in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
     }
 #define CHECK_CUDA_LAST()	{ \
     hipError_t ok = hipGetLastError(); \
     if ( ok != hipSuccess ) \
         fprintf(stderr, "-- Error CUDA last in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
     }
 
 
 /*
 Function initCentroids: This function copies the values of the initial centroids, using their
 position in the input data structure as a reference map.
 */
 void initCentroids(const float *data, float* centroids, const int* centroidPos, int samples, int K)
 {
     int i;
     int idx;
     for(i=0; i<K; i++)
     {
         idx = centroidPos[i];
         memcpy(&centroids[i*samples], &data[idx*samples], (samples*sizeof(float)));
     }
 }
 
 /*
 Function euclideanDistance: Euclidean distance
 This function could be modified
 */
 float euclideanDistance(const float *point, const float *center, int samples)
 {
     float dist=0.0;
     for(int i=0; i<samples; i++)
     {
         dist+= (point[i]-center[i])*(point[i]-center[i]);
     }
     dist = sqrt(dist);
     return(dist);
 }
 
 /*
 Function zeroFloatMatriz: Set matrix elements to 0
 This function could be modified
 */
 void zeroFloatMatriz(float *matrix, int rows, int columns)
 {
     int i,j;
     for (i=0; i<rows; i++)
         for (j=0; j<columns; j++)
             matrix[i*columns+j] = 0.0;
 }
 
 /*
 Function zeroIntArray: Set array elements to 0
 This function could be modified
 */
 void zeroIntArray(int *array, int size)
 {
     int i;
     for (i=0; i<size; i++)
         array[i] = 0;
 }
 
 __global__ void computeCentroids(const float* data, float* centroids, const int* classMap, int* pointsPerClass, int lines, int samples, int K) {
     int idx = blockIdx.x * blockDim.x + threadIdx.x;
     if (idx < lines)
     {
         int cluster = classMap[idx]-1;
         if (cluster >= 0 && cluster < K) {
             atomicAdd(&pointsPerClass[cluster], 1);
             for (int d = 0; d < samples; ++d) {
                 atomicAdd(&centroids[cluster * samples + d], data[idx * samples + d]);
             }
         }
     }
 }
 
 
 
 void kmeans(const char* input_filename, int K, int maxIterations,
             float minChangesPer, float maxThreshold, const char* output_filename) {
 
     //START CLOCK***************************************
     clock_t start, end;
     start = clock();
 
     // Reading the input data
     // lines = number of points; samples = number of dimensions per point
     int lines = 0, samples= 0;
 
     int error = readInput(input_filename, &lines, &samples);
     if(error != 0)
     {
         showFileError(error,input_filename);
         exit(error);
     }
 
     float *data = (float*)calloc(lines*samples,sizeof(float));
     if (data == NULL)
     {
         fprintf(stderr,"Memory allocation error.\n");
         exit(-4);
     }
     error = readInput2(input_filename, data);
     if(error != 0)
     {
         showFileError(error,input_filename);
         exit(error);
     }
 
     int minChanges= (int)(lines*minChangesPer/100.0);
     int *centroidPos = (int*)calloc(K,sizeof(int));
     float *centroids = (float*)calloc(K*samples,sizeof(float));
     int *classMap = (int*)calloc(lines,sizeof(int));
 
     if (centroidPos == NULL || centroids == NULL || classMap == NULL)
     {
         fprintf(stderr,"Memory allocation error.\n");
         exit(-4);
     }
 
     // Initial centroids
     srand(0);
     int i;
     for(i=0; i<K; i++)
         centroidPos[i]=rand()%lines;
 
     // Loading the array of initial centroids with the data from the array data
     // The centroids are points stored in the data array.
     initCentroids(data, centroids, centroidPos, samples, K);
 
 
     printf("\n\tData file: %s \n\tPoints: %d\n\tDimensions: %d\n", input_filename, lines, samples);
     printf("\tNumber of clusters: %d\n", K);
     printf("\tMaximum number of iterations: %d\n", maxIterations);
     printf("\tMinimum number of changes: %d [%g%% of %d points]\n", minChanges, minChangesPer, lines);
     printf("\tMaximum centroid precision: %f\n", maxThreshold);
 
     //END CLOCK*****************************************
     end = clock();
     printf("\nMemory allocation: %f seconds\n", (double)(end - start) / CLOCKS_PER_SEC);
     fflush(stdout);
 
     CHECK_CUDA_CALL( hipSetDevice(0) );
     CHECK_CUDA_CALL( hipDeviceSynchronize() );
     //**************************************************
     //START CLOCK***************************************
     start = clock();
     //**************************************************
     char *outputMsg = (char *)calloc(10000,sizeof(char));
     char line[100];
 
     int j;
     int clazz;
     float dist, minDist;
     int it=0;
     int changes = 0;
     float maxDist;
 
     //pointPerClass: number of points classified in each class
     //auxCentroids: mean of the points in each class
     int *pointsPerClass = (int *)malloc(K*sizeof(int));
     float *auxCentroids = (float*)malloc(K*samples*sizeof(float));
     float *distCentroids = (float*)malloc(K*sizeof(float));
     if (pointsPerClass == NULL || auxCentroids == NULL || distCentroids == NULL)
     {
         fprintf(stderr,"Memory allocation error.\n");
         exit(-4);
     }
 
 /*
  *
  * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
  *
  */
     float *d_data, *d_centroidSums;
     int *d_classMap, *d_pointsPerClass;
 
     CHECK_CUDA_CALL(hipMalloc(&d_data, lines * samples * sizeof(float)));
     CHECK_CUDA_CALL(hipMalloc(&d_centroidSums, K * samples * sizeof(float)));
     CHECK_CUDA_CALL(hipMalloc(&d_classMap, lines * sizeof(int)));
     CHECK_CUDA_CALL(hipMalloc(&d_pointsPerClass, K * sizeof(int)));
 
     CHECK_CUDA_CALL(hipMemcpy(d_data, data, lines * samples * sizeof(float), hipMemcpyHostToDevice));
 
     dim3 blockSize(256);
     dim3 gridSize((lines + blockSize.x - 1) / blockSize.x);
 
     do{
         it++;
 
         //1. Calculate the distance from each point to the centroid
         //Assign each point to the nearest centroid.
         changes = 0;
         for(i=0; i<lines; i++)
         {
             clazz=1;
             minDist=FLT_MAX;
             for(j=0; j<K; j++)
             {
                 dist=euclideanDistance(&data[i*samples], &centroids[j*samples], samples);
 
                 if(dist < minDist)
                 {
                     minDist=dist;
                     clazz=j+1;
                 }
             }
             if(classMap[i]!=clazz)
             {
                 changes++;
             }
             classMap[i]=clazz;
         }
 
         // 2. Recalculates the centroids: calculates the mean within each cluster
         CHECK_CUDA_CALL(hipMemcpy(d_classMap, classMap, lines * sizeof(int), hipMemcpyHostToDevice));
         
         CHECK_CUDA_CALL(hipMemset(d_centroidSums, 0, K * samples * sizeof(float)));
         CHECK_CUDA_CALL(hipMemset(d_pointsPerClass, 0, K * sizeof(int)));
         
         computeCentroids<<<gridSize, blockSize>>>(d_data, d_centroidSums, d_classMap, d_pointsPerClass, lines, samples, K);
         CHECK_CUDA_LAST();
         
         CHECK_CUDA_CALL(hipMemcpy(auxCentroids, d_centroidSums, K * samples * sizeof(float), hipMemcpyDeviceToHost));
         CHECK_CUDA_CALL(hipMemcpy(pointsPerClass, d_pointsPerClass, K * sizeof(int), hipMemcpyDeviceToHost))
 
         for(i=0; i<K; i++)
         {
             if (pointsPerClass[i] > 0) {
                 for(j=0; j<samples; j++){
                     auxCentroids[i*samples+j] /= pointsPerClass[i];
                 }
             }
         }
 
         maxDist=FLT_MIN;
         for(i=0; i<K; i++){
             distCentroids[i]=euclideanDistance(&centroids[i*samples], &auxCentroids[i*samples], samples);
             if(distCentroids[i]>maxDist) {
                 maxDist=distCentroids[i];
             }
         }
         memcpy(centroids, auxCentroids, (K*samples*sizeof(float)));
 
         sprintf(line,"\n[%d] Cluster changes: %d\tMax. centroid distance: %f", it, changes, maxDist);
         outputMsg = strcat(outputMsg,line);
 
     } while((changes>minChanges) && (it<maxIterations) && (maxDist>maxThreshold));
 
     hipFree(d_data);
     hipFree(d_centroidSums);
     hipFree(d_classMap);
     hipFree(d_pointsPerClass);
 
 /*
  *
  * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
  *
  */
     // Output and termination conditions
     printf("%s",outputMsg);
 
     CHECK_CUDA_CALL( hipDeviceSynchronize() );
 
     //END CLOCK*****************************************
     end = clock();
     printf("\nComputation: %f seconds", (double)(end - start) / CLOCKS_PER_SEC);
     fflush(stdout);
     //**************************************************
     //START CLOCK***************************************
     start = clock();
     //**************************************************
 
 
 
     if (changes <= minChanges) {
         printf("\n\nTermination condition:\nMinimum number of changes reached: %d [%d]", changes, minChanges);
     }
     else if (it >= maxIterations) {
         printf("\n\nTermination condition:\nMaximum number of iterations reached: %d [%d]", it, maxIterations);
     }
     else {
         printf("\n\nTermination condition:\nCentroid update precision reached: %g [%g]", maxDist, maxThreshold);
     }
 
     // Writing the classification of each point to the output file.
     error = writeResult(classMap, lines, output_filename);
     if(error != 0)
     {
         showFileError(error, output_filename);
         exit(error);
     }
 
     //Free memory
     free(data);
     free(classMap);
     free(centroidPos);
     free(centroids);
     free(distCentroids);
     free(pointsPerClass);
     free(auxCentroids);
 
     //END CLOCK*****************************************
     end = clock();
     printf("\n\nMemory deallocation: %f seconds\n", (double)(end - start) / CLOCKS_PER_SEC);
     fflush(stdout);
 
 
 }
 